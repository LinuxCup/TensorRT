#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <cstring>
#include <vector>
#include <hipcub/hipcub.hpp>

#include "NvInfer.h"
// #include "common/bertCommon.h"
#include "common/common.cuh"
#include "common/serialize.hpp"
// #include "geluPlugin.h"
#include "mysortPlugin.h"

// using namespace nvinfer1;
#define TOTAL_NUM (152064)
#define kGPUBlockSize (512)

namespace nvinfer1
{
namespace plugin
{

__global__ void find_objects_kernel(int total_num, int valid_num)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int i = iy + ix;
    if (i > valid_num - 1) return;

}

template<typename T>
void swap(T *a, T*b)
{
    T t= *a;
    *a = *b;
    *b = t;
}


int partition(int *arr, float *coor_array,  int l, int h)
{
    float x = coor_array[h];
    int i = l - 1;

    for(int j = l; j<= h-1; j++)
    {
        if(coor_array[j] >= x)
        {
            i++;
            swap(&coor_array[i],&coor_array[j]);
            swap(&arr[i],&arr[j]);
        }
    }
    swap(&coor_array[i+1],&coor_array[h]);
    swap(&arr[i+1],&arr[h]);
    return (i+1);
}

void quickSortIterative(int *arr, float *coor_array, int l, int h)
{
    // create an auxiliary stack
    int stack[10000];

    // init top of stack
    int top = -1;

    // push init values of l and h to stack
    stack[++top] = l;
    stack[++top] = h;

    while(top>=0)
    {
         h = stack[top--];
         l = stack[top--];
         
         int p = partition(arr,coor_array,l,h);

         if(p-1>l)
         {
            stack[++top] = l;
            stack[++top] = p -1;
         }

         if(p+1 <h)
         {
            stack[++top] = p+1;
            stack[++top] = h;
         }
    }
}

inline double getTime(void) {
        const auto t = std::chrono::system_clock::now();
        const auto t_sec = std::chrono::duration_cast<std::chrono::duration<double>>(t.time_since_epoch());
        return (double) t_sec.count();
    }

pluginStatus_t Mysort::sort_inference(hipStream_t stream, void const* const* inputs, void* const* outputs)
{
    double t0 = getTime();
    const void* src = inputs[0];
    const void* src_coor = inputs[1];
    const int* valid_num = (int*)inputs[2];
    void* ret = outputs[0];
    int valid_num_h;
    double t1 = getTime();
    hipMemcpyAsync(
            &valid_num_h,
            valid_num,
            sizeof(int),
            hipMemcpyDeviceToHost,
            stream
        );
    std::cout << "sort valid num: " << valid_num_h << std::endl;
    double t2 = getTime();
    

    // typedef hipcub::BlockRadixSort<int, 128, 512> BlockRadixSort;
    // __shared__ typename BlockRadixSort::TempStorage storageSort;
    hipMemcpyAsync(
            ret,
            src_coor,
            valid_num_h * sizeof(int),
            hipMemcpyDeviceToDevice,
            stream
        );

    double t3 = getTime();
    float* src_h = new float[valid_num_h];
    int* ret_h = new int[valid_num_h];
    double t4 = getTime();
    
    hipMemcpyAsync(
            src_h,
            src,
            valid_num_h * sizeof(float),
            hipMemcpyDeviceToHost,
            stream
        );
    hipMemcpyAsync(
            ret_h,
            ret,
            valid_num_h * sizeof(int),
            hipMemcpyDeviceToHost,
            stream
        );

    double t5 = getTime();
    quickSortIterative(ret_h, src_h, 0, valid_num_h-1);
    double t6 = getTime();

    // for (int i = 0; i < 50; i++)
    // {
    //     std::cout << ret_h[i] << " " << src_h[i] << std::endl;
    // }

    hipMemcpyAsync(
            ret,
            ret_h,
            valid_num_h * sizeof(int),
            hipMemcpyHostToDevice,
            stream
        );
    free(src_h);
    free(ret_h);
    double t7 = getTime();
    
    // int grid_size = (TOTAL_NUM + kGPUBlockSize - 1) / kGPUBlockSize;
    // find_objects_kernel<<<grid_size, kGPUBlockSize>>>(TOTAL_NUM, valid_num);

    // std::cout << "t0:" << t1 - t0 << std::endl;
    // std::cout << "t1:" << t2 - t1 << std::endl;
    // std::cout << "t2:" << t3 - t2 << std::endl;
    // std::cout << "t3:" << t4 - t3 << std::endl;
    // std::cout << "t4:" << t5 - t4 << std::endl;
    // std::cout << "t5:" << t6 - t5 << std::endl;
    // std::cout << "t6:" << t7 - t6 << std::endl;
    // std::cout << "total:" << t7 - t0 << std::endl;

    return STATUS_SUCCESS;
}





} // namespace plugin
} // namespace nvinfer1

